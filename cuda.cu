/*
 * Autores: Tomas Poblete Chamorro, Cristobal Perez Ramirez
 * Carrera: Ingeniería Civil Informática
 * Fecha: 10 de Junio de 2025
 * Propósito: Cálculo Paralelo con Cuda C
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define MAX 100
#define TOL 1e-6

// Kernel CUDA para transponer una matriz
__global__ void kernel_transpose(double* A, double* At, int m, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; // fila de A
    int j = blockIdx.x * blockDim.x + threadIdx.x; // columna de A
    if (i < m && j < n)
        At[j * m + i] = A[i * n + j];
}

// Kernel CUDA para multiplicar matrices
__global__ void kernel_mult(double* A, double* B, double* R, int r, int c, int p) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; // fila de A
    int j = blockIdx.x * blockDim.x + threadIdx.x; // columna de B
    if (i < r && j < p) {
        double suma = 0.0;
        for (int k = 0; k < c; k++)
            suma += A[i * c + k] * B[k * p + j];
        R[i * p + j] = suma;
    }
}

// Función CPU: invertir matriz cuadrada por sistemas lineales (puedes mejorarla con LAPACK)
int invertir(double A[MAX][MAX], double inv[MAX][MAX], int n) {
    double temp;
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++)
            inv[i][j] = (i == j) ? 1.0 : 0.0;

    for (int i = 0; i < n; i++) {
        if (fabs(A[i][i]) < TOL) return 0;
        temp = A[i][i];
        for (int j = 0; j < n; j++) {
            A[i][j] /= temp;
            inv[i][j] /= temp;
        }
        for (int k = 0; k < n; k++) {
            if (k == i) continue;
            temp = A[k][i];
            for (int j = 0; j < n; j++) {
                A[k][j] -= A[i][j] * temp;
                inv[k][j] -= inv[i][j] * temp;
            }
        }
    }
    return 1;
}

void leer_matriz(FILE* archivo, double A[MAX][MAX], int* m, int* n) {
    fscanf(archivo, "%d %d", m, n);
    for (int i = 0; i < *m; i++)
        for (int j = 0; j < *n; j++)
            fscanf(archivo, "%lf", &A[i][j]);
}

void escribir_salida(FILE* archivo, char tipo, double* R, int rows, int cols) {
    fprintf(archivo, "%c\n", tipo);
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            fprintf(archivo, "%.10lf", R[i * cols + j]);
            if (j < cols - 1) fprintf(archivo, " ");
        }
        fprintf(archivo, "\n");
    }
}

int main(int argc, char* argv[]) {
    // Configuración bloques e hilos por bloque, puedes parametrizarlo
    int hilosBloque = 16; // ejemplo, puedes variar por línea de comandos
    int bloques, bloquesY, bloquesX;
    double A[MAX][MAX], At[MAX][MAX], AA[MAX][MAX], AAt[MAX][MAX];
    double Inv[MAX][MAX], Resultado[MAX][MAX];
    int m, n;
    char archivo_entrada[128] = "entrada.ent";
    if (argc > 1) strcpy(archivo_entrada, argv[1]);
    FILE* entrada = fopen(archivo_entrada, "r");
    FILE* salida = fopen("salida.sal", "w");
    if (!entrada || !salida) {
        printf("Error abriendo archivos.\n");
        return 1;
    }

    leer_matriz(entrada, A, &m, &n);
    fclose(entrada);

    // Reserva y copia memoria a device
    double *d_A, *d_At, *d_AA, *d_AAt, *d_Resultado;
    size_t sz_A = m * n * sizeof(double);
    size_t sz_At = n * m * sizeof(double);
    size_t sz_AA = n * n * sizeof(double);
    size_t sz_AAt = m * m * sizeof(double);

    hipMalloc(&d_A, sz_A);
    hipMalloc(&d_At, sz_At);
    hipMemcpy(d_A, A, sz_A, hipMemcpyHostToDevice);

    // --- Transponer
    dim3 threadsTrans(hilosBloque, hilosBloque);
    dim3 gridTrans((n + hilosBloque - 1)/hilosBloque, (m + hilosBloque - 1)/hilosBloque);
    kernel_transpose<<<gridTrans, threadsTrans>>>(d_A, d_At, m, n);
    hipMemcpy(At, d_At, sz_At, hipMemcpyDeviceToHost);

    if (m <= n) {
        // Pseudoinversa derecha: R = A^T * (A * A^T)^-1
        hipMalloc(&d_AAt, sz_AAt);
        hipMalloc(&d_Resultado, sz_At);
        // Multiplica A * At
        dim3 gridMul((m + hilosBloque - 1)/hilosBloque, (m + hilosBloque - 1)/hilosBloque);
        kernel_mult<<<gridMul, threadsTrans>>>(d_A, d_At, d_AAt, m, n, m);
        hipMemcpy(AAt, d_AAt, sz_AAt, hipMemcpyDeviceToHost);
        if (!invertir(AAt, Inv, m)) {
            fprintf(salida, "-1\n");
            return 0;
        }
        // Multiplica At * Inv
        hipMemcpy(d_At, At, sz_At, hipMemcpyHostToDevice);
        hipMemcpy(d_AAt, Inv, sz_AAt, hipMemcpyHostToDevice);
        kernel_mult<<<gridTrans, threadsTrans>>>(d_At, d_AAt, d_Resultado, n, m, m);
        double resultado[MAX*MAX];
        hipMemcpy(resultado, d_Resultado, sz_At, hipMemcpyDeviceToHost);
        escribir_salida(salida, 'R', resultado, n, m);
    } else {
        // Pseudoinversa izquierda: L = (A^T * A)^-1 * A^T
        hipMalloc(&d_AA, sz_AA);
        hipMalloc(&d_Resultado, sz_At);
        // Multiplica At * A
        dim3 gridMul((n + hilosBloque - 1)/hilosBloque, (n + hilosBloque - 1)/hilosBloque);
        kernel_mult<<<gridMul, threadsTrans>>>(d_At, d_A, d_AA, n, m, n);
        hipMemcpy(AA, d_AA, sz_AA, hipMemcpyDeviceToHost);
        if (!invertir(AA, Inv, n)) {
            fprintf(salida, "-1\n");
            return 0;
        }
        hipMemcpy(d_AA, Inv, sz_AA, hipMemcpyHostToDevice);
        // Multiplica Inv * At
        kernel_mult<<<gridMul, threadsTrans>>>(d_AA, d_At, d_Resultado, n, n, m);
        double resultado[MAX*MAX];
        hipMemcpy(resultado, d_Resultado, sz_At, hipMemcpyDeviceToHost);
        escribir_salida(salida, 'L', resultado, n, m);
    }

    // Libera memoria
    hipFree(d_A); hipFree(d_At);
    hipFree(d_AA); hipFree(d_AAt); hipFree(d_Resultado);
    fclose(salida);
    return 0;
}